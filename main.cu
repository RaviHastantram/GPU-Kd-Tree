#include "hip/hip_runtime.h"
#include <iostream>

#include "kdtypes.h"
#include "geom.h"
#include "gpuBuilder.h"
#include "util.h"
#include "gpuTriangleList.h"
#include "gpuNode.h"
#include "cuPrintf.cuh"

using namespace std;

int main(int argc, char  ** argv)
{
	char * inputFile = argv[1];
	cudaPrintfInit();
	// load ply
	Mesh * m = loadMeshFromPLY(inputFile);
	
	// copy to device
	copyToGPU(m);
	
	GPUTriangleArray *d_triangleArray = new GPUTriangleArray();
	GPUNodeArray *d_nodeArray = new GPUNodeArray();

	int numActiveNodes=1;
	int numActiveTriangles=m->numTriangles;
	int threadsPerNode = 0;
	uint32 activeOffset;
	int numTotalNodes=1;
	uint32 numLeaves=0;
	uint32 currRound=0;

	// initialize device variables
	printf("initializeDeviceVariables\n");
	initializeDeviceVariables();

	
	// initialize the node list
	printf("initializeActiveNodeList\n");
	initializeActiveNodeList(d_nodeArray,d_triangleArray,m);

	while(numActiveNodes>0)
	{
		printf("Current round:%d\n",currRound);
		currRound++;

		printf("hipMemcpy\n");
		// copy offset to first active node to device
		HANDLE_ERROR( hipMemcpy(&d_activeOffset,&activeOffset,sizeof(uint32),hipMemcpyHostToDevice) );
		
		// calculate number of threads to assign to each node
		threadsPerNode = getThreadsPerNode(numActiveNodes,numActiveTriangles);
		
		printf("computeCost\n");
		// compute the split plane and value of each node
		computeCost <<< numActiveNodes,threadsPerNode >>>(d_nodeArray, d_triangleArray, d_nodeCounts, 
								d_triangleCounts, d_activeOffset, 
								d_triangles, d_points);
		CHECK_ERROR();

		cudaPrintfDisplay(stdout,true);
		CHECK_ERROR();

		printf("splitNodes\n");
		// split each node according to the plane and value chosen
		splitNodes<<<numActiveNodes,threadsPerNode>>>(d_nodeArray,d_triangleArray, d_nodeCounts, 
								d_triangleCounts, d_activeOffset,
								d_triangles, d_points);
		CHECK_ERROR();

		printf("hipDeviceSynchronize\n");
		// force threads to synchronize globally
		HANDLE_ERROR(hipDeviceSynchronize());
		
		printf("Update activeOffset\n");
		// increment pointer to first active node
		HANDLE_ERROR(hipMemcpy(&activeOffset,&d_activeOffset,sizeof(uint32),hipMemcpyDeviceToHost));
		activeOffset += numActiveNodes;
	
		printf("Count active nodes\n");
		// calculate number of active nodes in next round		
		numActiveNodes=countActiveNodes(numActiveNodes,d_numActiveNodes, d_nodeCounts);
		printf("numActiveNodes=%d\n",numActiveNodes);

		// update total nodes
		numTotalNodes += numActiveNodes;		
	
		printf("Count active triangles\n");
		// calculate number of triangles in next round
		numActiveTriangles=countActiveTriangles(numActiveNodes,d_numActiveTriangles, d_triangleCounts);		
	}

	// allocate host storage for nodes
	GPUNode * h_gpuNodes=new GPUNode[numTotalNodes];

	// copy out triangles out
	copyToHost(d_triangleArray, h_gpuNodes, &numLeaves, d_nodeArray->getNodes(), numTotalNodes);
	
	// copy triangles to disk
	dumpKDTree(h_gpuNodes, numTotalNodes, numLeaves,  m->bounds);
	cudaPrintfEnd();		
	return 0;
}
