#include "hip/hip_runtime.h"
#include "kdtypes.h"
#include "gpuBuilder.h"
#include <iostream>
#include <fstream>
#include <cstdio>
#include <cfloat>
#include <cassert>
#include "geom.h"

using namespace std;

///////////////////////////
// 
// Tree Building
//
///////////////////////////

uint32 getThreadsPerNode(int numActiveNodes,int numActiveTriangles)
{
	return 32;
}

__host__   void initializeActiveNodeList(GPUNodeArray* d_gpuNodes, GPUTriangleArray *d_triangleArray, Mesh * m)
{
	GPUNode h_node;
	h_node.nodeIdx=0;
	h_node.isLeaf=false;

	h_node.hostTriangles=new uint32[m->numTriangles];
	for(int i=0;i<m->numTriangles;i++)
	{
		h_node.hostTriangles[i]=i;
	}

	h_node.primBaseIdx=d_triangleArray->pushList(&h_node.hostTriangles,m->numTriangles);
	assert(h_node.primBaseIdx==0);

	h_node.primLength=m->numTriangles;

	h_node.nodeDepth=0;

	d_gpuNodes->pushNode(&h_node);
}

__device__ void computeCost(GPUNodeArray* d_gpuNodes, GPUTriangleArray* gpuTriangleList)
{
	__shared__ float mins[MAX_BLOCK_SIZE];
	__shared__ float maxs[MAX_BLOCK_SIZE];
	
	float min=FLT_MAX;
	float max=FLT_MIN;
 
	uint32 dim = blockIdx.x % 3;
	uint32 nodeIdx = blockIdx.x + d_activeOffset;
	GPUNode * node = d_gpuNodes->getNode(nodeIdx);

	if(node->nodeDepth>MAX_DEPTH)
	{
		node->splitChoice=SPLIT_NONE;
		node->isLeaf=true;
		return;
	}
	
	uint32 * triangleIDs = gpuTriangleList->getList(node->primBaseIdx);

	mins[threadIdx.x]=FLT_MAX;
	maxs[threadIdx.x]=FLT_MIN;

	uint32 currIdx = threadIdx.x;

	while(currIdx<node->primLength)
	{
		uint32 triangleID =  triangleIDs[currIdx];
		Triangle * triangle = &d_triangles[triangleID];
		for(uint32 j=0;j<3;j++)
		{
			uint32 pointID = triangle->ids[j];
			Point * point = &d_points[pointID];
			if(point->values[dim]<mins[threadIdx.x])
			{
				mins[threadIdx.x]=point->values[dim];
			}
			if(point->values[dim]>maxs[threadIdx.x])
			{
				maxs[threadIdx.x]=point->values[dim];
			}
		}
		currIdx += blockDim.x;
	}

	__syncthreads();

	if(threadIdx.x==0)
	{
		for(uint32 k=0;k<blockDim.x;k++)
		{
			if(mins[k]<min)
			{
				min=mins[k];
			}
			if(maxs[k]>max)
			{
				max=maxs[k];
			}
		}
		node->splitValue = 0.5*(min+max);
		node->splitChoice = dim;
	}
}

__device__ void splitNodes(GPUNodeArray* d_gpuNodes, GPUTriangleArray* gpuTriangleList, int * nodeCounts, int * triangleCounts)
{
	__shared__ uint32 offL[MAX_BLOCK_SIZE];
	__shared__ uint32 offD[MAX_BLOCK_SIZE];
	__shared__ uint32 offR[MAX_BLOCK_SIZE];
	__shared__ uint32 * leftList;
	__shared__ uint32 * rightList;
	__shared__ uint32 leftPrimBaseIdx;
	__shared__ uint32 rightPrimBaseIdx;

	uint32 triangleChoice;

	uint32 nodeIdx = blockIdx.x + d_activeOffset;
	GPUNode * node = d_gpuNodes->getNode(nodeIdx);
	int dim = node->splitChoice;
	float splitValue = node->splitValue;
	uint32 currIdx = threadIdx.x;
	uint32 * triangleIDs = gpuTriangleList->getList(node->primBaseIdx);
	uint32 leftBase=0, rightBase=0;
	uint32 leftCount=0, rightCount=0;

	if(threadIdx.x==0)
	{
		nodeCounts[blockIdx.x]=0;
		triangleCounts[blockIdx.x]=0;
	}
	
	if(node->isLeaf)
	{
		return;
	}

	if(threadIdx.x==0)
	{
		gpuTriangleList->lock();
		leftPrimBaseIdx=gpuTriangleList->allocateList(node->primLength);
		rightPrimBaseIdx=gpuTriangleList->allocateList(node->primLength);
		gpuTriangleList->unlock();

		leftList=gpuTriangleList->getList(leftPrimBaseIdx);
		rightList=gpuTriangleList->getList(rightPrimBaseIdx);
	}
	__syncthreads();

	float low = FLT_MIN;
	float high = FLT_MAX;
	
	//Need to initialize the offL, offD, offR arrays 
	while(currIdx<node->primLength)
	{
		offL[threadIdx.x]=0;
		offR[threadIdx.x]=0;
		offD[threadIdx.x]=0;

		uint32 triangleID =  triangleIDs[currIdx];
		Triangle * triangle = &d_triangles[triangleID];

		for(uint32 j=0;j<3;j++)
		{
			uint32 pointID = triangle->ids[j];
			Point * point = &d_points[pointID];
			if(point->values[dim]<low)
			{
				low=point->values[dim];
			}
			if(point->values[dim]>high)
			{
				high=point->values[dim];
			}
		}

		if( low < splitValue && high < splitValue )
		{
			offL[currIdx] = 1;
			triangleChoice=0;
		}

		if( low >= splitValue && high >= splitValue) 
		{
			offR[currIdx] = 1;
			triangleChoice=1;
		}

		if( low < splitValue && high >= splitValue ) 
		{
			offD[currIdx] = 1;
			triangleChoice=2;
		}

		__syncthreads();

		if(threadIdx.x==0)
		{
			for(uint32 k=1;k<blockDim.x;k++)
			{
				offL[k] += offL[k-1];
				offR[k] += offR[k-1];
				offD[k] += offD[k-1];
			}
			leftCount += offL[blockDim.x-1]+offD[blockDim.x-1];
			rightCount += offR[blockDim.x-1]+offD[blockDim.x-1];
		}

		__syncthreads();

		if(triangleChoice==0)
		{
			leftList[leftBase+offL[threadIdx.x]-1]=triangleID;
		}
		else if(triangleChoice==1)
		{
			rightList[rightBase+offR[threadIdx.x]-1]=triangleID;
		}
		else if(triangleChoice==2)
		{
			leftList[leftBase+offL[blockDim.x-1]+offD[threadIdx.x]-1]=triangleID;
			rightList[rightBase+offR[blockDim.x-1]+offD[threadIdx.x]-1]=triangleID;
		}
	
		leftBase += offL[blockDim.x-1]+offD[blockDim.x-1];
		rightBase += offR[blockDim.x-1]+offD[blockDim.x-1];

		currIdx += blockDim.x;
	}
	
	if(threadIdx.x==0)
	{
		d_gpuNodes->lock();

		GPUNode* leftNode =  d_gpuNodes->allocateNode();
		GPUNode* rightNode = d_gpuNodes->allocateNode();

		d_gpuNodes->unlock();

		node->leftIdx = leftNode->nodeIdx;
		node->rightIdx = rightNode->nodeIdx;

		leftNode->primBaseIdx=leftPrimBaseIdx;
		leftNode->primLength=leftCount;
		leftNode->nodeDepth=node->nodeDepth+1;
		
		rightNode->primBaseIdx=rightPrimBaseIdx;
		rightNode->primLength=rightCount;
		rightNode->nodeDepth=node->nodeDepth+1;
	
		nodeCounts[blockIdx.x]=1;
		triangleCounts[blockIdx.x]=leftCount+rightCount;
	}
}

////////////////////////////////
//
// Data Import/Export
//
///////////////////////////////
void copyToGPU(Mesh *mesh)
{
	//Copy the Points list
	int size = sizeof(Point)*(mesh->numPoints);
	hipMalloc(&d_points,size);
	hipMemcpy(d_points,mesh->points,size,hipMemcpyHostToDevice);

	//Copy the triangle list
	size = sizeof(Triangle)*(mesh->numTriangles);
	hipMalloc(&d_triangles, size);
	hipMemcpy(d_triangles,mesh->triangles,size,hipMemcpyHostToDevice);

	//Copy the mesh
	size = sizeof(Mesh);
	hipMalloc(&d_mesh,size);
	hipMemcpy(d_mesh,mesh,size,hipMemcpyHostToDevice);
}

void copyToHost(GPUTriangleArray * d_gpuTriangleArray, GPUNode * h_gpuNodes, uint32 * h_numLeaves, GPUNode * d_gpuNodes, uint32 numNodes)
{
	// copy the nodes
	hipMemcpy(h_gpuNodes,d_gpuNodes,sizeof(GPUNode)*numNodes,hipMemcpyDeviceToHost);
	for(int i=0;i<numNodes;i++)
	{
		GPUNode * node = &h_gpuNodes[i];
		if(node->isLeaf)
		{
			node->hostTriangles = new uint32[node->primLength];
			d_gpuTriangleArray->copyList(node->hostTriangles, node->primBaseIdx, node->primLength);
			*h_numLeaves++;
		}
	}
}

void dumpKDTree(GPUNode * nodes, uint32 numNodes, uint32 numLeaves, BoundingBox bounds)
{
	ofstream file("GPU-Kd-tree",ios::out | ios::binary);

	char *buffer = new char[100];
	
	unsigned int version = 1;

	//1. Write the LAYOUT_VERSION.
	file.write((char*)&version,sizeof(unsigned int));

	//2. Write the Bounds
	float zero=0;
	file.write((char*)&zero,sizeof(float));
	file.write((char*)&bounds.min,sizeof(float)*3);
	file.write((char*)&zero,sizeof(float));
	file.write((char*)&bounds.max,sizeof(float)*3);
	
	
	//3. Write the number of nodes.
	uint64_t n = (uint64_t)numNodes;
	file.write((char*)&n,sizeof(uint64_t));

	//4. Write the nodes.
	for(int i = 0; i < numNodes; i++)
	{
		dumpNode(file,i,nodes);		
	}

	//5.Write the number of leaves
	
	uint64_t leafCount = (uint64_t)numLeaves;
	file.write((char*)&leafCount,sizeof(uint64_t));

	//6. Write the triangles
	for(int i = 0; i < numNodes; i++)
	{
		if(nodes[i].isLeaf)
		{	
			dumpTriangles(file,i,nodes);
		}
	}

	file.close();
}


void dumpNode(ofstream& file,uint32 nodeID, GPUNode* nodes)
{
	GPUNode* node = &nodes[nodeID];
	uint32 data0 = 0;
	float data1 = 0; 
	if(node->isLeaf)
	{
		file.write((char*)&data0, sizeof(uint32));
		file.write((char*)&nodeID, sizeof(uint32));
	}
	else
	{
		data0 |= node->leftIdx;
		data0 <<= 2;
		data0 |= node->splitChoice;
		
		data1 = node->splitValue;
		
		file.write((char*)&data0, sizeof(uint32));
		file.write((char*)&data1, sizeof(float));
	}
}

void dumpTriangles(ofstream& file, uint32 nodeID, GPUNode* nodes)
{
	GPUNode* node = &nodes[nodeID];
	//7. Write the length of the triangle list
	uint64_t numTriangles = node->primLength;
	file.write((char*)&numTriangles,sizeof(uint64_t));

	//8. Write the triangles
	uint32 triangleIndex = 0; //index of the triangle in the PLY file
	for(int i = 0; i < numTriangles; i++)
	{
		triangleIndex = node->hostTriangles[i];
		file.write((char*)&triangleIndex,sizeof(triangleIndex));
	}
}



