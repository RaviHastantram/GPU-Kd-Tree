#include "hip/hip_runtime.h"
#include "kdtree.h"
#include "kdtypes.h"
#include "lists.h"
#include "gpuBuilder.h"
#include <iostream>
#include <fstream>
#include <cstdio>
#incluce <cfloat>

using namespace std;

///////////////////////////
// 
// Tree Building
//
///////////////////////////
uint32 getActiveNodes()
{
	uint32 numNodes=0;
	 
	if(hipMemcpyFromSymbol(&numNodes,
				HIP_SYMBOL("d_numActiveNodes"),
				sizeof(uint32),
				0,
				hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Copying d_numActiveNodes failed.\n");
	}
	return numNodes;
}

uint32 getThreadsPerNode(int numActiveNodes,int numActiveTriangles)
{
	return 0;
}

uint32 getActiveTriangles()
{
	uint32 numTriangles=0;
	
	if(hipMemcpyFromSymbol(&numTriangles,
				HIP_SYMBOL("d_numActiveTriangles"),
				sizeof(uint32),
				0,
				hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Copying d_numActiveTriangles failed.\n");
	}
	return numTriangles;
}

__device__ void computeCost()
{
	__shared__ float mins[MAX_BLOCK_SIZE];
	__shared__ float maxs[MAX_BLOCK_SIZE];
	
	uint32 min=FLT_MAX;
	uint32 max=FLT_MIN;
 
	uint32 dim = blockIdx.x % 3;
	uint32 nodeIdx = blockIdx.x + d_activeOffset;
	GPUNode * node = d_gpuNodes.getNode(nodeIdx);
	
	uint32 * triangleIDs= gpuTriangleList.getList(node->primBaseIdx);

	mins[threadIdx.x]=FLT_MAX;
	maxs[threadIdx.x]=FLT_MIN;

	uint32 currIdx = threadIdx.x;

	while(currIdx<node->primLength)
	{
		uint32 triangleID =  triangleIDs[currIdx];
		Triangle * triangle = d_triangles[triangleID];
		for(uint32 j=0;j<3;j++)
		{
			uint32 pointID = triangle->ids[j];
			Point * point = d_points[pointID];
			if(point->values[dim]<mins[threadIdx.x])
			{
				mins[threadIdx.x]=point->values[dim];
			}
			if(point->values[dim]>maxs[threadIdx.x])
			{
				maxs[threadIdx.x]=point->values[dim];
			}
		}
		currIdx += blockDim;
	}

	__syncthreads();

	if(threadIdx.x==0)
	{
		for(uint32 k=0;k<blockDim;k++)
		{
			if(mins[k]<min)
			{
				min=mins[k];
			}
			if(maxs[k]>max)
			{
				max=maxs[k];
			}
		}
		node->splitValue = 0.5*(min+max);
		node->splitChoice = dim;
	}
}

__device__ void splitNodes()
{
}

////////////////////////////////
//
// Data Import/Export
//
///////////////////////////////
void copyToGPU(Mesh *mesh)
{
	//Copy the Points list
	int size = sizeof(Point)*(mesh->numPoints);
	hipMalloc(&d_points,size);
	hipMemcpy(d_points,mesh->points,size,hipMemcpyHostToDevice);

	//Copy the triangle list
	size = sizeof(Triangle)*(mesh->numTriangles);
	hipMalloc(&d_triangles, size);
	hipMemcpy(d_triangles,mesh->triangles,size,hipMemcpyHostToDevice);

	//Copy the mesh
	size = sizeof(Mesh);
	hipMalloc(&d_mesh,size);
	hipMemcpy(d_mesh,mesh,size,hipMemcpyHostToDevice);
}




